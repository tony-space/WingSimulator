#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include "CudaLaunchHelpers.cuh"
#include "CDerivativeSolver.cuh"

using namespace wing2d::simulation::cuda;

static __device__ float2 SpringDamper(const float2& normal, const float2& vel1, const float2& vel2, float springLen)
{
	constexpr float stiffness = 10000.0f;
	constexpr float damp = 50.0f;
	auto v = dot(vel1 - vel2, normal);
	return normal * (springLen * stiffness + v * damp) * -1.0f;
}

static __device__ float2 SpringDamper2(const float2& normal, const float2& vel1, const float2& vel2, float springLen)
{
	constexpr float stiffness = 50000.0f;
	constexpr float damp = 50.0f;
	auto v = dot(vel1 - vel2, normal);
	return normal * (springLen * stiffness + v * damp) * -1.0f;
}

static __global__ void ParticleToWallKernel(const size_t particles, const float radius, const float2* __restrict__ pOdeState, SLineSegmentsSOA walls, float2* __restrict__ outForces)
{
	const auto threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= particles)
		return;

	const auto pos = pOdeState[threadId];
	const auto vel = pOdeState[threadId + particles];
	
	auto force = outForces[threadId];

	for (size_t i = 0; i < walls.lineSegments; ++i)
	{
		const auto d = walls.DistanceToLine(i, pos) - radius;
		if (d < 0.0f)
		{
			force += SpringDamper(walls.normal[i], vel, make_float2(0.0f), d);
		}
	}

	outForces[threadId] = force;
}

static __global__ void AddGravityKernel(float2* forces, unsigned n)
{
	const auto threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= n)
		return;

	forces[threadId].x += 0.5f;
}

static __global__ void BuildAirfoilBoxesKernel(SLineSegmentsSOA airfoil, SBoundingBoxesSOA boxes)
{
	const auto threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= airfoil.lineSegments)
		return;

	auto f = airfoil.first[threadId];
	auto s = airfoil.second[threadId];

	auto minCorner = make_float2(fminf(f.x, s.x), fminf(f.y, s.y));
	auto maxCorner = make_float2(fmaxf(f.x, s.x), fmaxf(f.y, s.y));

	boxes.min[threadId] = minCorner;
	boxes.max[threadId] = maxCorner;
}

static __global__ void BuildParticlesBoundingBoxesKernel(SBoundingBoxesSOA boundingBoxes, const float particleRad, const float2* __restrict__ particlePos)
{
	const auto threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= boundingBoxes.boundingBoxes)
		return;

	auto pos = particlePos[threadId];
	auto minCorner = make_float2(pos.x - particleRad, pos.y - particleRad);
	auto maxCorner = make_float2(pos.x + particleRad, pos.y + particleRad);

	boundingBoxes.min[threadId] = minCorner;
	boundingBoxes.max[threadId] = maxCorner;
}

CDerivativeSolver::CDerivativeSolver(size_t particles, float radius, const Segments_t& airfoil, const Segments_t& walls) :
	m_airfoilStorage(airfoil),
	m_wallsStorage(walls),
	m_airfoilsBoxesStorage(airfoil.size()),
	m_particlesBoxesStorage(particles),
	m_forces(particles),
	m_particles(particles),
	m_particleRad(radius)
{
	dim3 blockDim(kBlockSize);
	dim3 gridDim(GridSize(airfoil.size(), kBlockSize));
	auto boxesStorage = m_airfoilsBoxesStorage.get();
	BuildAirfoilBoxesKernel <<<gridDim, blockDim >>> (m_airfoilStorage.get(), boxesStorage);
	CudaCheckError();
	m_airfoilTree.Build(boxesStorage);
}

void CDerivativeSolver::Derive(const OdeState_t& curState, OdeState_t& outDerivative)
{
	ResetForces();
	BuildParticlesTree(curState);
	ResolveParticleParticleCollisions(curState);
	ResolveParticleWingCollisions(curState);
	ParticleToWall(curState);
	ApplyGravity();
	BuildDerivative(curState, outDerivative);
}

void CDerivativeSolver::ResetForces()
{
	auto devPtr = m_forces.data().get();
	auto bytesSize = m_particles * sizeof(decltype(m_forces)::value_type);
	CudaSafeCall(hipMemsetAsync(devPtr, 0, bytesSize));
}

void CDerivativeSolver::BuildParticlesTree(const OdeState_t& curState)
{
	dim3 blockDim(kBlockSize);
	dim3 gridDim(GridSize(m_particles, kBlockSize));


	auto boxesStorage = m_particlesBoxesStorage.get();

	BuildParticlesBoundingBoxesKernel <<<gridDim, blockDim >>>(boxesStorage, m_particleRad, curState.data().get());
	CudaCheckError();

	m_particlesTree.Build(boxesStorage);
}

void CDerivativeSolver::ResolveParticleParticleCollisions(const OdeState_t& curState)
{

}

void CDerivativeSolver::ResolveParticleWingCollisions(const OdeState_t& curState)
{

}

void CDerivativeSolver::ParticleToWall(const OdeState_t& curState)
{
	auto elements = unsigned(m_particles);
	dim3 blockDim(kBlockSize);
	dim3 gridDim((elements - 1) / blockDim.x + 1);

	ParticleToWallKernel <<<gridDim, blockDim >>> (m_particles, m_particleRad, curState.data().get(), m_wallsStorage.get(), m_forces.data().get());
	CudaCheckError();
}

void CDerivativeSolver::ApplyGravity()
{
	auto elements = unsigned(m_particles);
	dim3 blockDim(kBlockSize);
	dim3 gridDim((elements - 1) / blockDim.x + 1);

	AddGravityKernel <<<gridDim, blockDim >>> (m_forces.data().get(), elements);
	CudaCheckError();
}

void CDerivativeSolver::BuildDerivative(const OdeState_t& curState, OdeState_t& outDerivative) const
{
	const float2* d_velocities = curState.data().get() + m_particles;
	const float2* d_forces = m_forces.data().get();
	float2* d_derivative = outDerivative.data().get();

	const size_t dataBlockSize = m_particles * sizeof(float2);

	CudaSafeCall(hipMemcpyAsync(d_derivative, d_velocities, dataBlockSize, hipMemcpyKind::hipMemcpyDeviceToDevice));
	CudaSafeCall(hipMemcpyAsync(d_derivative + m_particles, d_forces, dataBlockSize, hipMemcpyKind::hipMemcpyDeviceToDevice));
}